#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "kernel.h"

void Acceleration_CPU(float* X, float* Y, float* AX, float* AY, int nt, int N, int id) {
	float ax = 0.f;
	float ay = 0.f;
	float xx, yy, rr;
	int sh = (nt - 1) * N;
	for (int j = 0; j < N; j++)
	{
		if (j != id) {
			xx = X[j + sh] - X[id + sh];
			yy = Y[j + sh] - Y[id + sh];
			rr = sqrtf(xx * xx + yy * yy);
			if (rr < 0.01f) {
				rr = 10.f / (rr * rr * rr);
				ax += xx * rr;
				ay += yy * rr;
			}
		}
	}
	AX[id] = ax;
	AY[id] = ay;
}

void Position_CPU(float* X, float* Y, float* VX, float* VY, float* AX, float* AY, float tau, int nt, int Np, int id) {
	int sh = (nt - 1) * Np;
	X[id + nt * Np] = X[id + sh] + VX[id] * tau + AX[id] * tau * tau * 0.5f;
	Y[id + nt * Np] = Y[id + sh] + VY[id] * tau + AY[id] * tau * tau * 0.5f;

	VX[id] = AX[id] * tau;
	VY[id] = AY[id] * tau;
}

int comparisonGPUAndCPU() {
	float timerValueGPU, timerValueCPU;
	hipEvent_t start, stop;
	hipEvent_t start1, stop1;
	hipEventCreate(&start);
	hipEventCreate(&start1);
	hipEventCreate(&stop);
	hipEventCreate(&stop1);

	int N = 10240; //число частиц
	int NT = 10; // число шагов
	float tau = 0.001f; // шаг по времени 0.001 с

	float* hX;
	float* hY;
	float* hVX;
	float* hVY;
	float* hAX;
	float* hAY;

	unsigned int mem_size = sizeof(float) * N;
	unsigned int mem_size_big = sizeof(float) * N * NT;

	hX = (float*)malloc(mem_size_big);
	hY = (float*)malloc(mem_size_big);
	hVX = (float*)malloc(mem_size);
	hVY = (float*)malloc(mem_size);
	hAX = (float*)malloc(mem_size);
	hAY = (float*)malloc(mem_size);

	float vv, phi;
	for (int j = 0; j < N; j++) {
		phi = (float)rand();
		hX[j] = rand() * cosf(phi) * 1.e-4f;
		hY[j] = rand() * sinf(phi) * 1.e-4f;
		vv = (hX[j] * hX[j] + hX[j] * hX[j]) * 10.f;
		hVX[j] = -vv * sinf(phi);
		hVY[j] = -vv * cosf(phi);
	}


	float* dX;
	float* dY;
	float* dVX;
	float* dVY;
	float* dAX;
	float* dAY;

	hipMalloc((void**)&dX, mem_size_big);
	hipMalloc((void**)&dY, mem_size_big);
	hipMalloc((void**)&dVX, mem_size);
	hipMalloc((void**)&dVY, mem_size);
	hipMalloc((void**)&dAX, mem_size);
	hipMalloc((void**)&dAY, mem_size);

	int N_thread = 256;
	int N_blocks = N / N_thread;

	

	hipMemcpy(dX, hX, mem_size_big, hipMemcpyHostToDevice);
	hipMemcpy(dY, hY, mem_size_big, hipMemcpyHostToDevice);
	hipMemcpy(dVX, hVX, mem_size, hipMemcpyHostToDevice);
	hipMemcpy(dVY, hVY, mem_size, hipMemcpyHostToDevice);

	hipEventRecord(start, 0);

	for (int j = 0; j < NT; j++) {
		Acceleration_GPU << < N_blocks, N_thread >> > (dX, dY, dAX, dAY, j, N);
		Position_GPU << < N_blocks, N_thread >> > (dX, dY, dVX, dVY, dAX, dAY, tau, j, N);		
	}

	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timerValueGPU, start, stop);

	hipMemcpy(hX, dX, mem_size_big, hipMemcpyDeviceToHost);
	hipMemcpy(hY, dY, mem_size_big, hipMemcpyDeviceToHost);


	
	

	for (int i = 0; i < N; i++)
	{
		printf("X[%d] = %.5f\n", i, hX[i]);
		printf("Y[%d] = %.5f\n", i, hY[i]);
	}

	hipEventRecord(start1, 0);

	int id;
	for (int j = 0; j < NT; j++) {
		for (id = 0; id < N; id++) {
			Acceleration_CPU(hX, hY, hAX, hAY, j, N, id);
			Position_CPU(hX, hY, hVX, hVY, hAX, hAY, tau, j, N, id);
		}
	}


	hipEventRecord(stop1, 0);
	hipEventSynchronize(stop1);
	hipEventElapsedTime(&timerValueCPU, start1, stop1);
	printf("\n GPU calculation time: %f ms\n", timerValueGPU);
	printf("\n CPU calculation time: %f ms\n", timerValueCPU);
	printf("\n Rate: %f x\n", timerValueGPU / timerValueCPU);


	free(hX);
	free(hY);
	free(hAX);
	free(hAY);
	free(hVX);
	free(hVY);

	hipFree(dX);
	hipFree(dVX);
	hipFree(dY);
	hipFree(dVY);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;

}

int main()
{
	return comparisonGPUAndCPU();
}


