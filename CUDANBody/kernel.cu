#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

__global__ void Acceleration_GPU(float* X, float* Y, float* AX, float* AY, int nt, int N)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	float ax = 0.f;
	float ay = 0.f;
	float xx, yy, rr;
	int sh = (nt - 1) * N;
	for (int j = 0; j < N; j++)
	{
		if (j != id) {
			xx = X[j + sh] - X[id + sh];
			yy = Y[j + sh] - Y[id + sh];
			rr = sqrtf(xx * xx + yy * yy);
			if (rr < 0.01f) {
				rr = 10.f / (rr * rr * rr);
				ax += xx * rr;
				ay += yy * rr;
			}
		}
	}
	AX[id] = ax;
	AY[id] = ay;
}

__global__ void Position_GPU(float* X, float* Y, float* VX, float* VY, float* AX, float* AY, float tau, int nt, int Np)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int sh = (nt - 1) * Np;
	X[id + nt * Np] = X[id + sh] + VX[id] * tau + AX[id] * tau * tau * 0.5f;
	Y[id + nt * Np] = Y[id + sh] + VY[id] * tau + AY[id] * tau * tau * 0.5f;

	VX[id] = AX[id] * tau;
	VY[id] = AY[id] * tau;
}